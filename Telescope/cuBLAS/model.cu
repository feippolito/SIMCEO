
#include "model.h"

#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>

/*
** Macros and support code before really 
** defining the main modules functionality.
*/

// Show the arrays in fashion manner
void showarray_device(type_c* x, int size) {
  cout << "[ ";
  for (int k = 0; k < size; k++)
    cout << *(x + k) << " ";
  cout << "]" << endl;
};

// Macro to transform row-major arrays to 
// column major arrays - C++ def to CUDA def
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

// Initialize the cuBLAS handle for further
// use.
hipblasHandle_t handle;


// Support methods

void Driver::Info(void) {
  cout << "Driver informations" << endl;
  // Show the dimensions
  cout << " * Dimensions..." << endl;
  cout << "   |- nx: " << this->hmodel.nx << " and " << this->dmodel.nx << endl;
  cout << "   |- nu: " << this->hmodel.nu << " and " << this->dmodel.nu << endl;
  cout << "   |- ny: " << this->hmodel.ny << " and " << this->dmodel.ny << endl;

  // Show the Host matrices
  cout << " * Host matrices..." << endl;
  cout << "   |- A: "; showarray_device(this->hmodel.A, this->hmodel.nx * this->hmodel.nx);
  cout << "   |- B: "; showarray_device(this->hmodel.B, this->hmodel.nx * this->hmodel.nu);
  cout << "   |- C: "; showarray_device(this->hmodel.C, this->hmodel.ny * this->hmodel.nx);
  
  // Show the Device matrices
  type_c* A_host; type_c* B_host; type_c* C_host;
  A_host = (type_c*)malloc(this->hmodel.b_size[0]);
  B_host = (type_c*)malloc(this->hmodel.b_size[1]);
  C_host = (type_c*)malloc(this->hmodel.b_size[2]);
  hipMemcpy(A_host, this->dmodel.A, this->hmodel.b_size[0], hipMemcpyDeviceToHost);
  hipMemcpy(B_host, this->dmodel.B, this->hmodel.b_size[1], hipMemcpyDeviceToHost);
  hipMemcpy(C_host, this->dmodel.C, this->hmodel.b_size[2], hipMemcpyDeviceToHost);
  cout << " * Device matrices..." << endl;
  cout << "   |- A: "; showarray_device(A_host, this->hmodel.nx * this->hmodel.nx);
  cout << "   |- B: "; showarray_device(B_host, this->hmodel.nx * this->hmodel.nu);
  cout << "   |- C: "; showarray_device(C_host, this->hmodel.nx * this->hmodel.ny);
};


// Main methods

void Driver::Init(int* dimensions, 
    type_c* A_matrix, type_c* B_matrix, 
    type_c* C_matrix, type_c* D_matrix ) {
  /*
  **  This is the initialization method of the cpp 
  ** fem library. It will define and initialize all
  ** host variables, such as model matrices and 
  ** simulation states.
  */

  // Set the host model dimensions
  this->hmodel.nx = dimensions[0];
  this->hmodel.nu = dimensions[1];
  this->hmodel.ny = dimensions[2];
  
  // Compute each matrix byte size
  this->hmodel.b_size = (type_i*)malloc(4*sizeof(type_i));
  this->hmodel.b_size[0] = this->hmodel.nx * this->hmodel.nx * sizeof(type_c);
  this->hmodel.b_size[1] = this->hmodel.nx * this->hmodel.nu * sizeof(type_c);
  this->hmodel.b_size[2] = this->hmodel.ny * this->hmodel.nx * sizeof(type_c);
  this->hmodel.b_size[3] = this->hmodel.ny * this->hmodel.nu * sizeof(type_c);

  // Allocate space for each model matrix
  this->hmodel.A = (type_c*)malloc(this->hmodel.b_size[0]);
  this->hmodel.B = (type_c*)malloc(this->hmodel.b_size[1]);
  this->hmodel.C = (type_c*)malloc(this->hmodel.b_size[2]);
  this->hmodel.D = (type_c*)malloc(this->hmodel.b_size[3]);
  
  // Attribute the values for each provided matrix in a
  // column major format, to be used in cuBLAS library
  for (int nx = 0; nx < this->hmodel.nx; nx++) {
    for (int nxi = 0; nxi < this->hmodel.nx; nxi++) 
      this->hmodel.A[IDX2C(nx,nxi,this->hmodel.nx)] = A_matrix[nx*this->hmodel.nx + nxi];
    for (int nu = 0; nu < this->hmodel.nu; nu++) 
      this->hmodel.B[IDX2C(nx,nu,this->hmodel.nx)] = B_matrix[nx*this->hmodel.nu + nu];
    for (int ny = 0; ny < this->hmodel.ny; ny++)
      this->hmodel.C[IDX2C(ny,nx,this->hmodel.ny)] = C_matrix[ny*this->hmodel.nx + nx];
  }
  
  // Compute each state byte size
  this->hstate.b_size = (type_i*)malloc(3*sizeof(type_i));
  this->hstate.b_size[0] = this->hmodel.nx * sizeof(type_c);
  this->hstate.b_size[1] = this->hmodel.nu * sizeof(type_c);
  this->hstate.b_size[2] = this->hmodel.ny * sizeof(type_c);
  
  // Allocate space for each state
  this->hstate.x_n = (type_c*)malloc(this->hstate.b_size[0]);
  this->hstate.x = (type_c*)malloc(this->hstate.b_size[0]);
  this->hstate.u = (type_c*)malloc(this->hstate.b_size[1]);
  this->hstate.y = (type_c*)malloc(this->hstate.b_size[2]);

  // Attribute the values for each state
  for (int k = 0; k < this->hmodel.nx; k++) {
    this->hstate.x[k] = 0.0;
    this->hstate.x_n[k] = 0.0;
  }
  for (int k = 0; k < this->hmodel.nu; k++)
    this->hstate.u[k] = 0.0;
  for (int k = 0; k < this->hmodel.ny; k++)
    this->hstate.y[k] = 0.0;

  cout << "Driver initialized..." << endl;
};

void Driver::Start(void) {
  /*
  ** Start all the device variables and respective
  ** values. It uses the already defined host variables
  ** to set all devices modules.
  */

  // Define the device dimensions
  this->dmodel.nx = this->hmodel.nx;
  this->dmodel.nu = this->hmodel.nu;
  this->dmodel.ny = this->hmodel.ny;

  // Define the device sizes
  this->dmodel.b_size = this->hmodel.b_size;
  this->dstate.b_size = this->hstate.b_size;

  // Allocate cuda memory for the model matrices
  hipMalloc(&this->dmodel.A, this->dmodel.b_size[0]);
  hipMalloc(&this->dmodel.B, this->dmodel.b_size[1]);
  hipMalloc(&this->dmodel.C, this->dmodel.b_size[2]);
  hipMalloc(&this->dmodel.D, this->dmodel.b_size[3]);

  // Attribute the device model matrices values 
  hipMemcpy(this->dmodel.A, this->hmodel.A, this->dmodel.b_size[0], hipMemcpyHostToDevice);
  hipMemcpy(this->dmodel.B, this->hmodel.B, this->dmodel.b_size[1], hipMemcpyHostToDevice);
  hipMemcpy(this->dmodel.C, this->hmodel.C, this->dmodel.b_size[2], hipMemcpyHostToDevice);
  hipMemcpy(this->dmodel.D, this->hmodel.D, this->dmodel.b_size[3], hipMemcpyHostToDevice);

  // Allocate cuda memory for the states
  hipMalloc(&this->dstate.x_n, this->dstate.b_size[0]);
  hipMalloc(&this->dstate.x, this->dstate.b_size[0]);
  hipMalloc(&this->dstate.u, this->dstate.b_size[1]);
  hipMalloc(&this->dstate.y, this->dstate.b_size[2]);

  // Attribute the device state values
  hipMemcpy(this->dstate.x_n, this->hstate.x_n, this->dstate.b_size[0], hipMemcpyHostToDevice);
  hipMemcpy(this->dstate.x, this->hstate.x, this->dstate.b_size[0], hipMemcpyHostToDevice);
  hipMemcpy(this->dstate.u, this->hstate.u, this->dstate.b_size[1], hipMemcpyHostToDevice);
  hipMemcpy(this->dstate.y, this->hstate.y, this->dstate.b_size[2], hipMemcpyHostToDevice);

  // Starting the cuBLAS computation module
  hipblasCreate(&handle);

  // Setting the enable and disable gains
  this->enable = 1.0; this->disable = 0.0;

  cout << "Driver started..." << endl;
};

type_c* Driver::Update(type_c* inputs) {
  /*
  **
  */

  hipMemcpy(this->dstate.u, inputs, this->dstate.b_size[1], hipMemcpyHostToDevice);

  // Compute: x(k+1) = (1 * A) * x(k) + 0 * x(k+1);
  //  |- Dimensions (m X n) * (n X k) = (m X k) => m, k, n
  //  |- Dimensions (nx X nx) * (nx X 1) = (nx X 1) => nx, 1, nx
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
    this->dmodel.nx, 1, this->dmodel.nx,
    &this->enable, this->dmodel.A, this->dmodel.nx, 
    this->dstate.x, this->dmodel.nx,
    &this->disable, this->dstate.x_n, this->dmodel.nx);
  
  // Compute: x(k+1) = (1 * B) * u(k) + 1 * x(k+1);
  //  |- Dimensions (m X n) * (n X k) = (m X k) => m, k, n
  //  |- Dimensions (nx X nu) * (nu X 1) = (nx X 1) => nx, 1, nu
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
    this->dmodel.nx, 1, this->dmodel.nu, 
    &this->enable, this->dmodel.B, this->dmodel.nx, 
    this->dstate.u, this->dmodel.nu,
    &this->enable, this->dstate.x_n, this->dmodel.nx);

  // Compute: x(k) = x(k+1)
  // Update the state variable...
  this->dstate.x = this->dstate.x_n;

  // Compute: y(k) = (1*C) * x(k) + 0 * y(k)
  //  |- Dimensions (m X n) * (n X k) = (m X k) => m, k, n
  //  |- Dimensions (ny X nx) * (nx X 1) = (ny X 1) => ny, 1, nx
  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
    this->dmodel.ny, 1, this->dmodel.nx,
    &this->enable, this->dmodel.C, this->dmodel.ny, 
    this->dstate.x, this->dmodel.nx,
    &this->disable, this->dstate.y, this->dmodel.ny);


};

type_c* Driver::Output(void) {
  /*
  ** Computes the output state on GPU, copies the 
  ** the values from device to host and returns the 
  ** type_c pointer with the output values.
  */

  
  // Copy values from device to host
  hipMemcpy(this->hstate.y, this->dstate.y, this->dstate.b_size[2], hipMemcpyDeviceToHost);

  return this->hstate.y;
}
