#define NPY_NO_DEPRECATED_API NPY_1_7_API_VERSION

#include "model.h"

#include <hip/hip_runtime.h>
#include <>
#include <hipsparse.h>

/*
** Macros and support code before really 
** defining the main modules functionality.
*/

// Show the arrays in fashion manner
void showarray_device(type_c* x, int size) {
  cout << "[ ";
  for (int k = 0; k < size; k++)
    cout << *(x + k) << " ";
  cout << "]" << endl;
};

void showarray_device2(type_i* x, int size) {
  cout << "[ ";
  for (int k = 0; k < size; k++)
    cout << *(x + k) << " ";
  cout << "]" << endl;
};

// Macro to transform row-major arrays to 
// column major arrays - C++ def to CUDA def
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

// Initialize the cuBLAS handle for further
// use.
hipsparseHandle_t handle;
hipsparseMatDescr_t descr;

// Support methods

void Driver::Info(void) {
  cout << "Driver informations" << endl;
  // Show the dimensions
  cout << " * Dimensions..." << endl;
  cout << "   |- nx: " << this->hmodel.nx << " and " << this->dmodel.nx << endl;
  cout << "   |- nu: " << this->hmodel.nu << " and " << this->dmodel.nu << endl;
  cout << "   |- ny: " << this->hmodel.ny << " and " << this->dmodel.ny << endl;

  // Show the Host matrices
  cout << " * Host matrices..." << endl;
  cout << "   |- A values: "; showarray_device(this->hmodel.A.values, this->hmodel.A.nnz);
  cout << "   |- A colInd: "; showarray_device2(this->hmodel.A.colInd, this->hmodel.A.nnz);
  cout << "   |- A rowPtr: "; showarray_device2(this->hmodel.A.rowPtr, this->hmodel.nx+1);
  cout << endl;
  
  cout << "   |- B values: "; showarray_device(this->hmodel.B.values, this->hmodel.B.nnz);
  cout << "   |- B colInd: "; showarray_device2(this->hmodel.B.colInd, this->hmodel.B.nnz);
  cout << "   |- B rowPtr: "; showarray_device2(this->hmodel.B.rowPtr, this->hmodel.nx+1);
  cout << endl;

  cout << "   |- C values: "; showarray_device(this->hmodel.C.values, this->hmodel.C.nnz);
  cout << "   |- C colInd: "; showarray_device2(this->hmodel.C.colInd, this->hmodel.C.nnz);
  cout << "   |- C rowPtr: "; showarray_device2(this->hmodel.C.rowPtr, this->hmodel.ny+1);
  cout << endl;

  // Show the Device matrices
  
  cout << " * Device matrices..." << endl;
  type_c* A_host_val; type_i* A_host_cInd; type_i* A_host_rPtr;
  A_host_val = (type_c*)malloc(this->hmodel.A.nnz*sizeof(type_c));
  A_host_cInd = (type_i*)malloc(this->hmodel.A.nnz*sizeof(type_i));
  A_host_rPtr = (type_i*)malloc((this->hmodel.nx+1)*sizeof(type_i));
  hipMemcpy(A_host_val, this->dmodel.A.values, this->dmodel.A.nnz*sizeof(type_c), hipMemcpyDeviceToHost);
  hipMemcpy(A_host_cInd, this->dmodel.A.colInd, this->dmodel.A.nnz*sizeof(type_i), hipMemcpyDeviceToHost);
  hipMemcpy(A_host_rPtr, this->dmodel.A.rowPtr, (this->dmodel.nx+1)*sizeof(type_i), hipMemcpyDeviceToHost);
  cout << "   |- A values: "; showarray_device(A_host_val, this->dmodel.A.nnz);
  cout << "   |- A colInd: "; showarray_device2(A_host_cInd, this->dmodel.A.nnz);
  cout << "   |- A rowPtr: "; showarray_device2(A_host_rPtr, this->dmodel.nx+1);
  cout << endl;
  
  type_c* B_host_val; type_i* B_host_cInd; type_i* B_host_rPtr;
  B_host_val = (type_c*)malloc(this->hmodel.B.nnz*sizeof(type_c));
  B_host_cInd = (type_i*)malloc(this->hmodel.B.nnz*sizeof(type_i));
  B_host_rPtr = (type_i*)malloc((this->hmodel.nx+1)*sizeof(type_i));
  hipMemcpy(B_host_val, this->dmodel.B.values, this->dmodel.B.nnz*sizeof(type_c), hipMemcpyDeviceToHost);
  hipMemcpy(B_host_cInd, this->dmodel.B.colInd, this->dmodel.B.nnz*sizeof(type_i), hipMemcpyDeviceToHost);
  hipMemcpy(B_host_rPtr, this->dmodel.B.rowPtr, (this->dmodel.nx+1)*sizeof(type_i), hipMemcpyDeviceToHost);
  cout << "   |- B values: "; showarray_device(B_host_val, this->dmodel.B.nnz);
  cout << "   |- B colInd: "; showarray_device2(B_host_cInd, this->dmodel.B.nnz);
  cout << "   |- B rowPtr: "; showarray_device2(B_host_rPtr, this->dmodel.nx+1);
  cout << endl;

  type_c* C_host_val; type_i* C_host_cInd; type_i* C_host_rPtr;
  C_host_cInd = (type_i*)malloc(this->hmodel.C.nnz*sizeof(type_i));
  C_host_rPtr = (type_i*)malloc((this->hmodel.ny+1)*sizeof(type_i));
  C_host_val = (type_c*)malloc(this->hmodel.C.nnz*sizeof(type_c));
  hipMemcpy(C_host_val, this->dmodel.C.values, this->dmodel.C.nnz*sizeof(type_c), hipMemcpyDeviceToHost);
  hipMemcpy(C_host_cInd, this->dmodel.C.colInd, this->dmodel.C.nnz*sizeof(type_i), hipMemcpyDeviceToHost);
  hipMemcpy(C_host_rPtr, this->dmodel.C.rowPtr, (this->dmodel.ny+1)*sizeof(type_i), hipMemcpyDeviceToHost);
  cout << "   |- C values: "; showarray_device(C_host_val, this->dmodel.C.nnz);
  cout << "   |- C colInd: "; showarray_device2(C_host_cInd, this->dmodel.C.nnz);
  cout << "   |- C rowPtr: "; showarray_device2(C_host_rPtr, this->dmodel.ny+1);
  cout << endl;
};


void Driver::buildHostSparseModel(type_c* A_matrix, type_c* B_matrix, 
    type_c* C_matrix, type_c* D_matrix) {
  
  // Build the host State Space model 
  this->hdense2csr(A_matrix, 0);     //
  this->hdense2csr(B_matrix, 1);     //
  this->hdense2csr(C_matrix, 2);     //
  this->hdense2csr(D_matrix, 3);     //

  // Build the host states as dense
  // Compute each state byte size
  this->hstate.b_size = (type_i*)malloc(3*sizeof(type_i));
  this->hstate.b_size[0] = this->hmodel.nx * sizeof(type_c);
  this->hstate.b_size[1] = this->hmodel.nu * sizeof(type_c);
  this->hstate.b_size[2] = this->hmodel.ny * sizeof(type_c);
  
  // Allocate space for each state
  this->hstate.x_n = (type_c*)malloc(this->hstate.b_size[0]);
  this->hstate.x = (type_c*)malloc(this->hstate.b_size[0]);
  this->hstate.u = (type_c*)malloc(this->hstate.b_size[1]);
  this->hstate.y = (type_c*)malloc(this->hstate.b_size[2]);

  // Attribute the values for each state
  for (int k = 0; k < this->hmodel.nu; k++)
    this->hstate.u[k] = 0.0;
  for (int k = 0; k < this->hmodel.ny; k++)
    this->hstate.y[k] = 0.0;
  for (int k = 0; k < this->hmodel.nx; k++) {
    this->hstate.x[k]   = 0.0;
    this->hstate.x_n[k] = 0.0;
  }
};

void Driver::hdense2csr(type_c* matrix, type_i type) {
  /*
  **
  */

  // ---- A matrix ----
  if ( type == 0 ) {
    // Count the non zero values 
    this->hmodel.A.nnz = 2 * this->hmodel.nx;
    // Allocate memory for the csr parameters
    this->hmodel.A.values = (type_c*)malloc(this->hmodel.A.nnz*sizeof(type_c));
    this->hmodel.A.rowPtr = (type_i*)malloc((this->hmodel.nx+1)*sizeof(type_i));
    this->hmodel.A.colInd = (type_i*)malloc(this->hmodel.A.nnz*sizeof(type_i));
    // Attribute the csr values
    int item_count = 0; int col_index = 0;
    for (int k = 0; k < this->hmodel.nx; k++) {
      // Get the start column index
      col_index = k;
      if (k >= this->hmodel.nx/2)
        col_index = col_index - this->hmodel.nx/2;
      // Define the row start pointer
      this->hmodel.A.rowPtr[k] = item_count;
      // Add the first value
      this->hmodel.A.values[item_count] = matrix[k*this->hmodel.nx+col_index];
      this->hmodel.A.colInd[item_count] = col_index;
      item_count += 1;
      // Add the second value
      col_index = col_index + this->hmodel.nx/2;
      this->hmodel.A.values[item_count] = matrix[k*this->hmodel.nx+col_index];
      this->hmodel.A.colInd[item_count] = col_index;
      item_count += 1;
    }
    this->hmodel.A.rowPtr[this->hmodel.nx] = this->hmodel.A.nnz;
  }

  // ---- B matrix ----
  if ( type == 1 ) {
    // Count the non zero values
    this->hmodel.B.nnz = this->hmodel.nx * this->hmodel.nu / 2;
    // Allocate memory for the csr parameters
    this->hmodel.B.values = (type_c*)malloc(this->hmodel.B.nnz*sizeof(type_c));
    this->hmodel.B.rowPtr = (type_i*)malloc((this->hmodel.nx+1)*sizeof(type_i));
    this->hmodel.B.colInd = (type_i*)malloc(this->hmodel.B.nnz*sizeof(type_i));
    // Attribute the csr values
    int item_count = 0; int col_index = 0;
    for (int k = 0; k < this->hmodel.nx; k++) {
      if ( k >= this->hmodel.nx/2 ) {
        // Define the pointer to where this 
        // row starts in the values
        this->hmodel.B.rowPtr[k] = item_count;
        // Include values on the csr matrix
        for (int j = 0; j < this->hmodel.nu; j++) {
          // Add the new value to the csr values
          // attribute and define the column id
          this->hmodel.B.values[item_count] = matrix[k*this->hmodel.nu+j];
          this->hmodel.B.colInd[item_count] = j;
          item_count += 1;
        }
      } else {
        this->hmodel.B.rowPtr[k] = 0;
      }
    }
    this->hmodel.B.rowPtr[this->hmodel.nx] = this->hmodel.B.nnz;
  }

  // ---- C matrix ----
  if ( type == 2 ) {
    // Count the non zero values
    this->hmodel.C.nnz = this->hmodel.ny * this->hmodel.nx / 2;
    // Allocate memory for the csr parameters
    this->hmodel.C.values = (type_c*)malloc(this->hmodel.C.nnz*sizeof(type_c));
    this->hmodel.C.rowPtr = (type_i*)malloc((this->hmodel.ny+1)*sizeof(type_i));
    this->hmodel.C.colInd = (type_i*)malloc(this->hmodel.C.nnz*sizeof(type_i));
    // Attribute the csr values
    int item_count = 0; int col_index = 0;
    for (int k = 0; k < this->hmodel.ny; k++) {
      // Define the initialization pointer 
      // for this row
      this->hmodel.C.rowPtr[k] = item_count;
      // Include values on the csr matrix
      for (int j = 0; j < this->hmodel.nx/2; j++) {
        // Add the new value to the csr values
        // attribute and define the column id
        this->hmodel.C.values[item_count] = matrix[k*this->hmodel.nx+j];
        this->hmodel.C.colInd[item_count] = j;
        item_count += 1;
      }
    }
    this->hmodel.C.rowPtr[this->hmodel.ny] = this->hmodel.C.nnz;
  }
};

void Driver::buildDeviceSparseModel(void) {
  /*
  **
  */

  // Build the device State Space model
  // --- A matrix ---
  this->dmodel.A.nnz = this->hmodel.A.nnz;
  // Allocate the device memory
  hipMalloc(&this->dmodel.A.values, this->dmodel.A.nnz*sizeof(type_c));
  hipMalloc(&this->dmodel.A.colInd, this->dmodel.A.nnz*sizeof(type_i));
  hipMalloc(&this->dmodel.A.rowPtr, (this->dmodel.nx+1)*sizeof(type_i));
  // Copy csr attributes values to the device
  hipMemcpy(this->dmodel.A.values, this->hmodel.A.values, 
    this->dmodel.A.nnz*sizeof(type_c), hipMemcpyHostToDevice);
  hipMemcpy(this->dmodel.A.colInd, this->hmodel.A.colInd, 
    this->dmodel.A.nnz*sizeof(type_i), hipMemcpyHostToDevice);
  hipMemcpy(this->dmodel.A.rowPtr, this->hmodel.A.rowPtr, 
    (this->dmodel.nx+1)*sizeof(type_i), hipMemcpyHostToDevice);

  // --- B matrix ---
  this->dmodel.B.nnz = this->hmodel.B.nnz;
  // Allocate the device memory
  hipMalloc(&this->dmodel.B.values, this->dmodel.B.nnz*sizeof(type_c));
  hipMalloc(&this->dmodel.B.colInd, this->dmodel.B.nnz*sizeof(type_i));
  hipMalloc(&this->dmodel.B.rowPtr, (this->dmodel.nx+1)*sizeof(type_i));
  // Copy csr attributes values to the device
  hipMemcpy(this->dmodel.B.values, this->hmodel.B.values, 
    this->dmodel.B.nnz*sizeof(type_c), hipMemcpyHostToDevice);
  hipMemcpy(this->dmodel.B.colInd, this->hmodel.B.colInd, 
    this->dmodel.B.nnz*sizeof(type_i), hipMemcpyHostToDevice);
  hipMemcpy(this->dmodel.B.rowPtr, this->hmodel.B.rowPtr, 
    (this->dmodel.nx+1)*sizeof(type_i), hipMemcpyHostToDevice);

  // --- C matrix ---
  this->dmodel.C.nnz = this->hmodel.C.nnz;
  // Allocate the device memory

  hipMalloc(&this->dmodel.C.values, this->dmodel.C.nnz*sizeof(type_c));
  hipMalloc(&this->dmodel.C.colInd, this->dmodel.C.nnz*sizeof(type_i));
  hipMalloc(&this->dmodel.C.rowPtr, (this->dmodel.ny+1)*sizeof(type_i));
  // // Copy csr attributes values to the device
  hipMemcpy(this->dmodel.C.values, this->hmodel.C.values, 
    this->dmodel.C.nnz*sizeof(type_c), hipMemcpyHostToDevice);
  hipMemcpy(this->dmodel.C.colInd, this->hmodel.C.colInd, 
    this->dmodel.C.nnz*sizeof(type_i), hipMemcpyHostToDevice);
  hipMemcpy(this->dmodel.C.rowPtr, this->hmodel.C.rowPtr, 
    (this->dmodel.ny+1)*sizeof(type_i), hipMemcpyHostToDevice);

  // Build the device states as dense
  this->dstate.b_size = this->hstate.b_size;
  // Allocate cuda memory for the states
  hipMalloc(&this->dstate.x_n, this->dstate.b_size[0]);
  hipMalloc(&this->dstate.x, this->dstate.b_size[0]);
  hipMalloc(&this->dstate.u, this->dstate.b_size[1]);
  hipMalloc(&this->dstate.y, this->dstate.b_size[2]);

  // Attribute the device state values
  hipMemcpy(this->dstate.x_n, this->hstate.x_n, this->dstate.b_size[0], hipMemcpyHostToDevice);
  hipMemcpy(this->dstate.x, this->hstate.x, this->dstate.b_size[0], hipMemcpyHostToDevice);
  hipMemcpy(this->dstate.u, this->hstate.u, this->dstate.b_size[1], hipMemcpyHostToDevice);
  hipMemcpy(this->dstate.y, this->hstate.y, this->dstate.b_size[2], hipMemcpyHostToDevice);
}

void Driver::driverCleanUp(void) {
  /*
  **
  */

  // Clean the device/host model matrices
  // --- A matrix ---
  hipFree(this->dmodel.A.values);
  hipFree(this->dmodel.A.colInd);
  hipFree(this->dmodel.A.rowPtr);
  free(this->hmodel.A.values);
  free(this->hmodel.A.colInd);
  free(this->hmodel.A.rowPtr);

  // --- B matrix ---
  hipFree(this->dmodel.B.values);
  hipFree(this->dmodel.B.colInd);
  hipFree(this->dmodel.B.rowPtr);
  free(this->hmodel.B.values);
  free(this->hmodel.B.colInd);
  free(this->hmodel.B.rowPtr);

  // --- C matrix ---
  hipFree(this->dmodel.C.values);
  hipFree(this->dmodel.C.colInd);
  hipFree(this->dmodel.C.rowPtr);
  free(this->hmodel.C.values);
  free(this->hmodel.C.colInd);
  free(this->hmodel.C.rowPtr);

  // Clean the device/host states
  hipFree(this->dstate.x_n);
  free(this->hstate.x_n);
  hipFree(this->dstate.x);
  free(this->hstate.x);
  hipFree(this->dstate.u);
  free(this->hstate.u);
  hipFree(this->dstate.y);
  free(this->hstate.y);

  cout << "All memory spaces freed..." << endl;
  cout << "Driver terminated." << endl;
}

// Main methods

void Driver::Init(int* dimensions, 
    type_c* A_matrix, type_c* B_matrix, 
    type_c* C_matrix, type_c* D_matrix ) {
  /*
  **  This is the initialization method of the cpp 
  ** fem library. It will define and initialize all
  ** host variables, such as model matrices and 
  ** simulation states.
  */

  // Set the host/device model dimensions
  this->hmodel.nx = dimensions[0];  this->dmodel.nx = dimensions[0];
  this->hmodel.nu = dimensions[1];  this->dmodel.nu = dimensions[1];
  this->hmodel.ny = dimensions[2];  this->dmodel.ny = dimensions[2];

  // Setup host simulation environment
  this->buildHostSparseModel(A_matrix, B_matrix, C_matrix, D_matrix);
  this->buildDeviceSparseModel();

  cout << "Driver initialized..." << endl;
};

void Driver::Start(void) {
  /*
  ** Start all the device variables and respective
  ** values. It uses the already defined host variables
  ** to set all devices modules.
  */
  
  // Starting the cuBLAS computation module
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

  // Setting the enable and disable gains
  this->enable = 1.0; this->disable = 0.0;

  cout << "Driver started..." << endl;
};

void Driver::Update(type_c* inputs) {
  /*
  **
  */

  hipMemcpy(this->dstate.u, inputs, this->dstate.b_size[1], hipMemcpyHostToDevice);

  // Compute: x(k+1) = (1 * A) * x(k) + 0 * x(k+1);
  //  |- Dimensions (m X n) * (n X k) = (m X k) => m, k, n
  //  |- Dimensions (nx X nx) * (nx X 1) = (nx X 1) => nx, 1, nx
  hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, 
    this->dmodel.nx, this->dmodel.nx, this->dmodel.A.nnz,
    &this->enable, descr, 
    this->dmodel.A.values, this->dmodel.A.rowPtr, this->dmodel.A.colInd,
    this->dstate.x, 
    &this->disable, this->dstate.x_n);
  // hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
  //   this->dmodel.nx, 1, this->dmodel.nx,
  //   &this->enable, this->dmodel.A, this->dmodel.nx, 
  //   this->dstate.x, this->dmodel.nx,
  //   &this->disable, this->dstate.x_n, this->dmodel.nx);
  
  // Compute: x(k+1) = (1 * B) * u(k) + 1 * x(k+1);
  //  |- Dimensions (m X n) * (n X k) = (m X k) => m, k, n
  //  |- Dimensions (nx X nu) * (nu X 1) = (nx X 1) => nx, 1, nu
  hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, 
    this->dmodel.nx, this->dmodel.nu, this->dmodel.B.nnz,
    &this->enable, descr, 
    this->dmodel.B.values, this->dmodel.B.rowPtr, this->dmodel.B.colInd,
    this->dstate.u, 
    &this->enable, this->dstate.x_n);
  // hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
  //   this->dmodel.nx, 1, this->dmodel.nu, 
  //   &this->enable, this->dmodel.B, this->dmodel.nx, 
  //   this->dstate.u, this->dmodel.nu,
  //   &this->enable, this->dstate.x_n, this->dmodel.nx);

  // Compute: x(k) = x(k+1)
  // Update the state variable...
  this->dstate.x = this->dstate.x_n;

  // Compute: y(k) = (1*C) * x(k) + 0 * y(k)
  //  |- Dimensions (m X n) * (n X k) = (m X k) => m, k, n
  //  |- Dimensions (ny X nx) * (nx X 1) = (ny X 1) => ny, 1, nx
  hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, 
    this->dmodel.ny, this->dmodel.nx, this->dmodel.C.nnz,
    &this->enable, descr, 
    this->dmodel.C.values, this->dmodel.C.rowPtr, this->dmodel.C.colInd,
    this->dstate.x, 
    &this->disable, this->dstate.y);
  // hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
  //   this->dmodel.ny, 1, this->dmodel.nx,
  //   &this->enable, this->dmodel.C, this->dmodel.ny, 
  //   this->dstate.x, this->dmodel.nx,
  //   &this->disable, this->dstate.y, this->dmodel.ny);

};

type_c* Driver::Output(void) {
  /*
  ** Computes the output state on GPU, copies the 
  ** the values from device to host and returns the 
  ** type_c pointer with the output values.
  */

  // Copy values from device to host
  // hipMemcpy(this->hstate.x_n, this->dstate.x_n, this->dstate.b_size[0], hipMemcpyDeviceToHost);
  hipMemcpy(this->hstate.y, this->dstate.y, this->dstate.b_size[2], hipMemcpyDeviceToHost);

  return this->hstate.y;
}

void Driver::Terminate(void) {
  /*
  **
  */
  this->driverCleanUp();
}
